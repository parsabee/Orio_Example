#include "hip/hip_runtime.h"
/*
TC:64
BC:28
SC:2
CB:True
PL:16
CFLAGS:
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define N 1000
double *y;
double *x;
void malloc_arrays() {
  
  y = (double*) malloc((N) * sizeof(double));
  x = (double*) malloc((N) * sizeof(double));
}
void init_input_vars() {
  int i1;
  for (i1=0; i1<N; i1++)
   y[i1] = (i1) % 5 + 1;
  for (i1=0; i1<N; i1++)
   x[i1] = (i1) % 5 + 1;
}

double orio_t_start, orio_t_end, orio_t = (double)LONG_MAX;



__global__ void orcu_kernel2260(const int n, double* y, double* x) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[64];
  __shared__ double shared_x[64];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_y[threadIdx.x]=shared_y[threadIdx.x]+shared_x[threadIdx.x];
    y[i]=shared_y[threadIdx.x];
  }
}


int main(int argc, char *argv[]) {
  
#ifdef MAIN_DECLARATIONS
  MAIN_DECLARATIONS()
#endif
  malloc_arrays();
  init_input_vars();

  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  float orcu_elapsed=0.0, orcu_transfer=0.0;
  hipEvent_t tstart, tstop, start, stop;
  hipEventCreate(&tstart); hipEventCreate(&tstop);
  hipEventCreate(&start);  hipEventCreate(&stop);
  
  for (int orio_i=0; orio_i<ORIO_REPS; orio_i++) {
    
    
    

  int n=N;

  /*@ begin Loop(transform CUDA(threadCount=TC, blockCount=BC, streamCount=SC, cacheBlocks=CB, preferL1Size=PL)

  for (i=0; i<=n-1; i++)
    y[i]+=x[i];

  ) @*/
  {
    hipDeviceSynchronize();
    /*declare variables*/
    double* dev_y;
    double* dev_x;
    int nthreads=64;
    int nstreams=2;
    /*calculate device dimensions*/
    dim3 dimGrid, dimBlock;
    dimBlock.x=nthreads;
    dimGrid.x=28;
    /*create streams*/
    int istream, soffset;
    hipStream_t stream[nstreams+1];
    for (istream=0; istream<=nstreams; istream++ ) 
      hipStreamCreate(&stream[istream]);
    int chunklen=n/nstreams;
    int chunkrem=n%nstreams;
    /*allocate device memory*/
    hipMalloc(&dev_y,N*sizeof(double));
    hipHostRegister(y,N*sizeof(double),hipHostRegisterPortable);
    hipMalloc(&dev_x,N*sizeof(double));
    hipHostRegister(x,N*sizeof(double),hipHostRegisterPortable);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    /*copy data from host to device*/
    hipEventRecord(tstart,0);
    for (istream=0; istream<nstreams; istream++ ) {
      soffset=istream*chunklen;
      hipMemcpyAsync(dev_y+soffset,y+soffset,chunklen*sizeof(double),hipMemcpyHostToDevice,stream[istream]);
      hipMemcpyAsync(dev_x+soffset,x+soffset,chunklen*sizeof(double),hipMemcpyHostToDevice,stream[istream]);
    }
    if (chunkrem!=0) {
      soffset=istream*chunklen;
      hipMemcpyAsync(dev_y+soffset,y+soffset,chunkrem*sizeof(double),hipMemcpyHostToDevice,stream[istream]);
      hipMemcpyAsync(dev_x+soffset,x+soffset,chunkrem*sizeof(double),hipMemcpyHostToDevice,stream[istream]);
    }
    hipEventRecord(tstop,0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&orcu_transfer,tstart,tstop);
    hipEventRecord(start,0);
    /*invoke device kernel*/
    int blks4chunk=dimGrid.x/nstreams;
    if (dimGrid.x%nstreams!=0) 
      blks4chunk++ ;
    for (istream=0; istream<nstreams; istream++ ) {
      soffset=istream*chunklen;
      orcu_kernel2260<<<blks4chunk,dimBlock,0,stream[istream]>>>(chunklen,dev_y+soffset,dev_x+soffset);
    }
    if (chunkrem!=0) {
      soffset=istream*chunklen;
      orcu_kernel2260<<<blks4chunk,dimBlock,0,stream[istream]>>>(chunkrem,dev_y+soffset,dev_x+soffset);
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&orcu_elapsed,start,stop);
    /*copy data from device to host*/
    for (istream=0; istream<nstreams; istream++ ) {
      soffset=istream*chunklen;
      hipMemcpyAsync(y+soffset,dev_y+soffset,chunklen*sizeof(double),hipMemcpyDeviceToHost,stream[istream]);
    }
    if (chunkrem!=0) {
      soffset=istream*chunklen;
      hipMemcpyAsync(y+soffset,dev_y+soffset,chunkrem*sizeof(double),hipMemcpyDeviceToHost,stream[istream]);
    }
    for (istream=0; istream<=nstreams; istream++ ) 
      hipStreamSynchronize(stream[istream]);
    hipDeviceSetCacheConfig(hipFuncCachePreferNone);
    for (istream=0; istream<=nstreams; istream++ ) 
      hipStreamDestroy(stream[istream]);
    /*free allocated memory*/
    hipFree(dev_y);
    hipFree(dev_x);
    hipHostUnregister(y);
    hipHostUnregister(x);
    hipError_t err=hipGetLastError();
    if (hipSuccess!=err) 
      printf("CUDA runtime error: %s@",hipGetErrorString(err));
  }
  /*@ end @*/
  

    
    printf("{'[1, 1, 1, 0, 0, 0]' : (%g,%g)}\n", orcu_elapsed, orcu_transfer);
  }
  
  hipEventDestroy(tstart); hipEventDestroy(tstop);
  hipEventDestroy(start);  hipEventDestroy(stop);
  
  
  return 0;
}
